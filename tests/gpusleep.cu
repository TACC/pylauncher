// -*- c++ -*-
/****************************************************************
 *
 * This example program is part of the pylauncher distribution
 * copyright Victor Eijkhout 2020
 *
 * Usage: gpusleep t [ tmax ]
 * -- if tmax not given: random sleep up to `t' seconds
 * -- if tmax given    : random sleep time in [t,tmax] interval
 *
 ****************************************************************/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>

// Kernel function to print "Hello, World!"
// __host__ __device__
__global__
 void helloWorldKernel() {
#if defined(__CUDA_ARCH__)
    printf("Hello from device\n");
#else
    printf("Hello from host\n");
#endif
};

// __global__ void helloWorldKernel() {
//   printf("Hello, World!\n");
// }

int main(int argc,char **argv) {
  srand(time(NULL));

  int tmin,tmax;

  switch (argc) {
  case 1 : 
    printf("Usage: randomaction t [ tmax ]\n"); 
    tmin = 4; tmax = 6;
    break;
  case 2 : 
    tmin = atoi(argv[1]); tmax = tmin;
    break;
  case 3 : 
    tmin = atoi(argv[1]); tmax = atoi(argv[2]);
    break;
  default:
    printf("Usage: randomaction t [ tmax ]\n"); 
    break;
  }

  int nseconds;
  if (tmin==tmax)
    nseconds = tmin;
  else 
    nseconds = tmin + rand() % (tmax-tmin);

  printf("I am going to sleep for %d seconds\n",nseconds);
//  helloWorldKernel();
  helloWorldKernel<<<1, 1>>>();
  // Wait for the GPU to finish executing the kernel
  hipDeviceSynchronize();
  sleep(nseconds);
  printf(" .. done kernel and back\n");

  return 0;
}
